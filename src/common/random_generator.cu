#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file random_generator.cu
 * \brief gpu functions for random number generator.
 */

#include <algorithm>
#include <mshadow/cuda/tensor_gpu-inl.cuh>
#include "./random_generator.h"
#include "./cuda_utils.h"

namespace mxnet {
namespace common {

__global__ void rand_generator_seed_kernel(RandGenerator<gpu> *pgen, unsigned int seed) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  pgen->Seed(seed, id);
}

template<>
inline void RandGeneratorSeed(RandGenerator<gpu> *gen, unsigned int seed) {
  using namespace mshadow::cuda;
  int ngrid = std::min(kMaxGridNum, (CURAND_STATE_SIZE + kBaseThreadNum - 1) / kBaseThreadNum);
  rand_generator_seed_kernel<<<ngrid, kBaseThreadNum, 0, 0>>>(gen, seed);
}

template<>
inline RandGenerator<gpu> *NewRandGenerator() {
  RandGenerator<gpu> *gen;
  CUDA_CALL(hipMalloc(&gen, sizeof(RandGenerator<gpu>)));
  return gen;
};

}  // namespace common
}  // namespace mxnet