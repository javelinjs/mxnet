#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file random_generator.cu
 * \brief gpu functions for random number generator.
 */

#include <algorithm>
#include "./random_generator.h"
#include "../operator/mxnet_op.h"
#include "./cuda_utils.h"

namespace mxnet {
namespace common {
namespace random {

template<typename DType>
__global__ void rand_generator_seed_kernel(RandGenerator<gpu, DType> *pgen, unsigned int seed) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  pgen->Seed(seed, id);
}

template<>
inline void RandGeneratorSeed<gpu, float>(RandGenerator<gpu, DType> *gen, unsigned int seed) {
  using namespace mshadow::cuda;
  int ngrid = std::min(kMaxGridNum, (CURAND_STATE_SIZE + kBaseThreadNum - 1) / kBaseThreadNum);
  rand_generator_seed_kernel<<<ngrid, kBaseThreadNum, 0, 0>>>(gen, seed);
}

template<>
RandGenerator<gpu, float> *NewRandGenerator<gpu, float>() {
  RandGenerator<gpu, float> *gen;
  CUDA_CALL(hipMalloc(&gen, sizeof(RandGenerator<gpu, float>)));
  return gen;
};

template<>
inline void DeleteRandGenerator(RandGenerator<gpu, float> *p) {
  if (p) hipFree(p);
}

}  // namespace random
}  // namespace common
}  // namespace mxnet